#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <ctime>
#include <stdio.h>
#include <vector>
#include <limits>
#include <algorithm>

#include <Windows.h>

hipError_t sortWithCuda(int *a, size_t size, float* time);

typedef long long int64;
typedef unsigned long long uint64;
__host__ int64 GetTimeMs64()
{
	/* Windows */
	FILETIME ft;
	LARGE_INTEGER li;

	GetSystemTimeAsFileTime(&ft);
	li.LowPart = ft.dwLowDateTime;
	li.HighPart = ft.dwHighDateTime;

	uint64 ret = li.QuadPart;
	ret -= 116444736000000000LL;

	ret /= 10000; 


	return ret;
}


// Sortowanie w GPU
__global__ void swapOnKernel(int *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x * 2;
	int cacheFirst;
	int cacheSecond;
	int cacheThird;

	for (int j = 0; j < size / 2 + 1; j++) {

		if (i + 1 < size) {
			cacheFirst = a[i];
			cacheSecond = a[i + 1];

			if (cacheFirst > cacheSecond) {
				int temp = cacheFirst;
				a[i] = cacheSecond;
				cacheSecond = a[i + 1] = temp;
			}
		}

		if (i + 2 < size) {
			cacheThird = a[i + 2];
			if (cacheSecond > cacheThird) {
				int temp = cacheSecond;
				a[i + 1] = cacheThird;
				a[i + 2] = temp;
			}
		}
		//Synchronizacja w�tk�w
		__syncthreads();
	}

}

//Klasyczny bubble sort na CPU
__host__ void bubbleSort(int arr[], int n) {
	// �r�d�o http://www.algolist.net/Algorithms/Sorting/Bubble_sort
	bool swapped = true;
	int j = 0;
	int tmp;
	while (swapped) {
		swapped = false;
		j++;
		for (int i = 0; i < n - j; i++) {
			if (arr[i] > arr[i + 1]) {
				tmp = arr[i];
				arr[i] = arr[i + 1];
				arr[i + 1] = tmp;
				swapped = true;
			}
		}
	}
}

int main()
{
	srand((unsigned)time(0));
	const int arraySize = 2000;

	//utworzenie wektora (jednowymiarowej tablicy) i zape�nienie go losowymi warto�ciami
	std::vector<int> a(arraySize);
	for (int i = 0; i < arraySize; ++i) {
		a[i] = (rand() * (rand() - 1));
	}
	std::vector<int> b(a);

	float time = 0.0;

	// wywo�anie sortowania na GPU po�rednio poprzez funkcj� hosta
	hipError_t cudaStatus = sortWithCuda(&a[0], a.size(), &time);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "sortWithCuda failed!");
		return 1;
	}

	bool sortingSuccessful = true;
	for (int i = 0; i < a.size() - 1; ++i) {
		if (a[i] > a[i + 1]) {
			sortingSuccessful = false;
			break;
		}
		 printf("%d, ", a[i]);
	}
	printf("\n");

	printf("Time for the GPU: %f ms\n", time);

	if (!sortingSuccessful) {
		printf("Sorting failed.\n");
	}
	//uruchomienie sortowania na CPU i zmierzenie czasu
	int64 stlSortStart = GetTimeMs64();
	bubbleSort(&b[0], b.size());
	int64 stlSortFinish = GetTimeMs64();
	printf("Time for the CPU: %d ms\n",
		(stlSortFinish - stlSortStart));


//zwolnienie zasob�w
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//getchar();

	return 0;
}

//uruchomienie kernela z ca�� otoczk�
__host__ hipError_t sortWithCuda(int *a, size_t size, float* time)
{
	int *dev_a = 0;
	hipError_t cudaStatus;

	// sprawdzenie czy mamy kompatybiln� kart� graficzn�
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// alokacja pami�ci
	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// kopiowanie z hosta na kart�
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// zrobienie tajmera
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//wystartowanie tajmera
	hipEventRecord(start, 0);
	// uruchomineie sortowania na GPU
	swapOnKernel << <1, size / 2 >> > (dev_a, size);
	// Zatrzymanie tajmera i obliczenie czasu trwania oblicze�
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(time, start, stop);

	// kopiowanie wyniku do osta
	cudaStatus = hipMemcpy(a, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_a);

	return cudaStatus;
}